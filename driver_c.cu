#include "hip/hip_runtime.h"
#include <stdio.h>
#include "common.h"

__global__ void times(int *d_nums, int t);
extern "C" void driver_c_(int *nums, int *m, int *n);

extern "C" void driver_c_(int *nums, int *m, int *n) {
  // 1D stuff
//  dim3 grid( 1 );
//  dim3 block( *n ); 

//  const int arr_size = *n*sizeof(int);

//  int *d_nums;
//  CHECK(hipMalloc((void **)&d_nums, arr_size));
//
//  CHECK(hipMemcpy(d_nums, nums, arr_size, hipMemcpyHostToDevice));
//  times<<<grid, block>>>(d_nums, 2);
//  CHECK(hipMemcpy(nums, d_nums, arr_size, hipMemcpyDeviceToHost));
//
//  CHECK(hipFree(d_nums));

  printf("nums[0]: \n", nums[0]);

}

__global__
void times(int *d_nums, int t) {
  int i = threadIdx.x;
  d_nums[i] = t*d_nums[i];
}
